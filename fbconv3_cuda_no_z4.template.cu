#include "hip/hip_runtime.h"
#raw
#include <stdio.h>
#end raw

#set NF=4
#set Z=4
#set xyzw = ['x','y','z','w']


texture<float4, 1, hipReadModeElementType> tex_float4;
__constant__ float constant[$Z][$N_FILTER_ROWS][$FILTER_W][$N_OUTPUT4S][$NF];

#define uint unsigned int

#if $IMUL_FAST
#define IMUL(a, b) __mul24(a, b)
#else
#define IMUL(a, b) a*b
#end if


extern "C" {

#for nk in xrange($N_KERNELS)

  __global__
  void cudafilter_kernel_${nk}
  (
   float4 *input
#for o in xrange($N_OUTPUT4S)
   , float4 *output$o
#end for
   )
  {

#if $PAD_SHARED_IN
    __shared__ float shared_in[$BLOCK_H][$N_FILTER_ROWS][$Z][$INPUT_BLOCK_W+1];
#else
    __shared__ float shared_in[$BLOCK_H][$N_FILTER_ROWS][$Z][$INPUT_BLOCK_W];
#end if

    // -- input/output "pointers"
    const uint in_idx =				   \
      IMUL(IMUL(blockIdx.y, $BLOCK_H), $INPUT_W) + \
      IMUL(IMUL($nk, $INPUT_W), $N_FILTER_ROWS) +  \
      IMUL(threadIdx.y, $INPUT_W) +		   \
      IMUL(blockIdx.x, $BLOCK_W) + threadIdx.x ;
    
    const uint out_idx =				\
      IMUL(IMUL(blockIdx.y, $BLOCK_H), $OUTPUT_W) +	\
      IMUL(threadIdx.y, $OUTPUT_W) +			\
      IMUL(blockIdx.x, $BLOCK_W) + threadIdx.x ;
    
    // -- XXX
    float4 input_v4;

    // -------------------------------------------------------------------------
    // -- load input to shared memory
    // -------------------------------------------------------------------------
#for nfr in xrange($N_FILTER_ROWS)
#for i in xrange($N_LOAD_ITERATIONS)
#if $i==($N_LOAD_ITERATIONS-1)
    if( (threadIdx.x + IMUL($BLOCK_W, $i)) < $INPUT_BLOCK_W )
#end if
      {
	input_v4 = tex1Dfetch(tex_float4, in_idx + IMUL($INPUT_W, $nfr) + IMUL($BLOCK_W, $i));
#for d in xrange($NF)
	shared_in[threadIdx.y][$nfr][$d][threadIdx.x + IMUL($BLOCK_W, $i)] = input_v4.$xyzw[$d];
#end for
      }
#end for
#end for
    __syncthreads();

    // -------------------------------------------------------------------------
    // -- compute dot products
    // -------------------------------------------------------------------------
    float value, weight;

#for o in xrange($N_OUTPUT4S)
#for n in xrange($NF)
    float sum${o}${n} = 0;
#end for
#end for

#for d in xrange($Z)
#for nfr in xrange($N_FILTER_ROWS)
#for i in xrange($FILTER_W)
    value = shared_in[threadIdx.y][$nfr][$d][threadIdx.x+$i];
#for o in xrange($N_OUTPUT4S)
#for n in xrange($NF)
    weight = constant[$d][$nfr][$i][$o][$n];
    sum${o}${n} += value*weight;
#end for
#end for
#end for
#end for
#end for


    // -------------------------------------------------------------------------
    // -- output results
    // -------------------------------------------------------------------------

/*     output[s_out_idx[threadIdx.y][threadIdx.x]].x += sum0; */
#for o in xrange($N_OUTPUT4S)
#for n in xrange($NF)
    output${o}[out_idx].$xyzw[$n] += sum${o}${n};
#end for
#end for

  }
#end for



}

